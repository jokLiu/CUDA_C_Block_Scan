#include "hip/hip_runtime.h"
//=========================================================================================
//							DETAILS ABOUT THE SUBMISSION
//=========================================================================================
// Name: Jokubas Liutkus
// ID: 1601768
//
// Goals Achieved (all goals were achieved):
//	1. block scan +
//	2. full scan for large vectors +
//	3. Bank conflict avoidance optimization (BCAO) +
//
// Your time, in milliseconds to execute the different scans on a vector of 10,000,000 entries:
//	∗ Block scan without BCAO:
//	∗ Block scan with BCAO:
//	∗ Full scan without BCAO:
//	∗ Full scan with BCAO:
//
// CPU model: Intel(R) Core(TM) i5-6500 CPU @ 3.20GHz x 4 (lab machine)
// GPU model: GeForce GTX 960 (lab machine)
//
// Improvements/Additional features:
//	1. All the multiplications were replaced with bit shifting which improved the performance
//	2. Extract sum phase was merged with the main block scan function removing additional kernel call
//	3. Padded the shared memory array of the last block to zero while loading in the data from device memory
//	   to get the faster operations on the last block elements.
//	4. Saving the last element of each block to the local variable from the shared memory
//	   before running reduction and distribution phases rather than loading it later from a global memory
//
//=========================================================================================
//=========================================================================================
//=========================================================================================

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <math.h>

// A helper macro to simplify handling cuda error checking
#define CUDA_ERROR( err, msg ) { \
if (err != hipSuccess) {\
    printf( "%s: %s in %s at line %d\n", msg, hipGetErrorString( err ), __FILE__, __LINE__);\
    exit( EXIT_FAILURE );\
}\
}

// Block size and its double version
#define BLOCK_SIZE 512
#define BLOCK_SIZE_TWICE BLOCK_SIZE*2

// for avoiding bank conflicts
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
		((n) >> NUM_BANKS + (n) >> (LOG_NUM_BANKS << 1))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif


static void compare_results(const int *vector1, const int *vector2,
		int numElements) {
	for (int i = 0; i < numElements; ++i) {
//		printf("%d ----------- %d\n", vector1[i], vector2[i]);
		if (fabs(vector1[i] - vector2[i]) > 1e-5f) {
			printf("%d ----------- %d\n", vector1[i], vector2[i]);
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}
}

__host__
void sequential_scan(int *g_idata, int *g_odata, int n) {

	g_odata[0] = 0;
	for (int i = 1; i < n; i++) {
		g_odata[i] = g_odata[i - 1] + g_idata[i - 1];
	}
}

__global__
void add_to_block(int *block, int len_block, int *SUM) {
	int s = SUM[blockIdx.x];

	int addr = blockIdx.x * (blockDim.x << 1) + threadIdx.x;

	__syncthreads();
	if (addr < len_block)
		block[addr] += s;
	if (addr + blockDim.x < len_block)
		block[addr + blockDim.x] += s; // TODO need to check properly

}

__global__
void block_scan_full_BCAO(int *g_idata, int *g_odata, int n, int *SUM,
		int add_last) {

	const int blockSize = (BLOCK_SIZE << 1);

	__shared__ int temp[blockSize + (BLOCK_SIZE / 8)]; // allocated on invocation

	int thid = threadIdx.x;
	int thid_shift = thid << 1;
	int blockId = blockIdx.x * (BLOCK_SIZE << 1);
	int offset = 0;  //1;
	int last = 0;

	int ai = thid;
	int bi = thid + BLOCK_SIZE;  //(n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	if (blockId + ai < n) {
		temp[ai + bankOffsetA] = g_idata[blockId + ai];
	} else
		temp[ai + bankOffsetA] = 0;
	if (blockId + bi < n) {
		temp[bi + bankOffsetB] = g_idata[blockId + bi];
	} else
		temp[bi + bankOffsetB] = 0;

	if (add_last && thid == BLOCK_SIZE - 1) // save the last element for later
		last =
				temp[blockSize - 1 + CONFLICT_FREE_OFFSET((BLOCK_SIZE << 1) - 1)];

	for (int d = BLOCK_SIZE ; d > 0; d >>= 1) // build sum in place up the tree
			{
		__syncthreads();
		if (thid < d) {
			int ai = ((thid_shift + 1) << offset) - 1;
			int bi = ((thid_shift + 2) << offset) - 1;

			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			temp[bi] += temp[ai];
		}

		offset++;
	}

	if (thid == 0) {
		temp[blockSize - 1 + CONFLICT_FREE_OFFSET(blockSize - 1)] = 0;
	}

	for (int d = 1; d < blockSize; d <<= 1) // traverse down tree & build scan
			{
		offset--;
		__syncthreads();
		if (thid < d) {
			int ai = ((thid_shift + 1) << offset) - 1;
			int bi = ((thid_shift + 2) << offset) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();
	if (add_last && thid == BLOCK_SIZE - 1) // save the last element for later
		SUM[blockIdx.x] = temp[blockSize - 1
				+ CONFLICT_FREE_OFFSET(blockSize - 1)] + last;
	if (blockId + ai < n)
		g_odata[blockId + ai] = temp[ai + bankOffsetA];
	if (blockId + bi < n)
		g_odata[blockId + bi] = temp[bi + bankOffsetB];
}

__host__
void full_block_scan_BCAO(int *h_IN, int *h_OUT, int len) {

	// -----------------------------------------------------------
	// 							INITIALIZATION
	// -----------------------------------------------------------

	// error code to check return values for CUDA class
	hipError_t err = hipSuccess;

	// size to allocate for the vectors
	size_t size = len * sizeof(int);

	// create device timer
	hipEvent_t d_start, d_stop;
	float d_msecs;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);

	// allocate memory for all the possible vectors needed for the execution
	int *d_IN = NULL;
	err = hipMalloc((void **) &d_IN, size);
	CUDA_ERROR(err, "Failed to allocate device vector IN");

	int *d_OUT = NULL;
	err = hipMalloc((void**) &d_OUT, size);
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_1 = NULL;
	err = hipMalloc((void**) &d_SUM_1,
			(1 + ((len - 1) / (BLOCK_SIZE * 2))) * sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_1_Scanned = NULL;
	err = hipMalloc((void**) &d_SUM_1_Scanned,
			(1 + ((len - 1) / (BLOCK_SIZE * 2))) * sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_2 = NULL;
	err = hipMalloc((void**) &d_SUM_2, (BLOCK_SIZE << 1) * sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_2_Scanned = NULL;
	err = hipMalloc((void**) &d_SUM_2_Scanned,
			(BLOCK_SIZE << 1) * sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	// copy the memory from the host to the device
	err = hipMemcpy(d_IN, h_IN, size, hipMemcpyHostToDevice);
	CUDA_ERROR(err, "Failed to copy array IN from host to device");

	// size of the grid for each level
	int blocksPerGridLevel1 = 1 + ((len - 1) / (BLOCK_SIZE * 2));
	int blocksPerGridLevel2 = 1 + ceil(blocksPerGridLevel1 / (BLOCK_SIZE << 1));
	int blocksPerGridLevel3 = 1 + ceil(blocksPerGridLevel2 / (BLOCK_SIZE << 1));

	// -----------------------------------------------------------
	// 							EXECUTION
	// -----------------------------------------------------------

	// choosing the level on which to run the kernels
	// based on the size of the grids

	// if level one grid size is equal to 1 then single
	// LEVEL 1 is enough to scan the whole array
	if (blocksPerGridLevel1 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);
		printf("lanuched1!\n");
		// execute the actual kernel
		block_scan_full_BCAO<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
				len,
				NULL, 0);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if level two grid size is equal to 1 then two (LEVEL 2)
	// scans are required to scan the whole array
	else if (blocksPerGridLevel2 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);
		printf("lanuched2!\n");
		// execute the actual kernels
		printf("%d\n", blocksPerGridLevel1);
		block_scan_full_BCAO<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
				len, d_SUM_1, 1);
		block_scan_full_BCAO<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1,
				d_SUM_1_Scanned, blocksPerGridLevel1, NULL, 0);
		add_to_block<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_OUT, len,
				d_SUM_1_Scanned);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if level 3 grid size is equal to 1 then three (LEVEL 3)
	// scans are required to scan the whole array
	else if (blocksPerGridLevel3 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);
		printf("lanuched3!\n");
		// execute the actual kernels
		block_scan_full_BCAO<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
				len, d_SUM_1, 1);
		block_scan_full_BCAO<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1,
				d_SUM_1_Scanned, blocksPerGridLevel1, d_SUM_2, 1);
		block_scan_full_BCAO<<<1, BLOCK_SIZE>>>(d_SUM_2, d_SUM_2_Scanned,
				blocksPerGridLevel2, NULL, 0);
		add_to_block<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1_Scanned,
				blocksPerGridLevel1, d_SUM_2_Scanned);
		add_to_block<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_OUT, len,
				d_SUM_1_Scanned);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if none of the conditions above is met, it means that the array is too
	// large to be scanned in 3 level scan, therefore we print the error message
	// and return
	else {
		fprintf(stderr,
				"The array size=%d is too large to be scanned with level 3 scan!\n",
				len);

		// using goto is discouraged, however, in such situations
		// where in the error conditions exit or cleanup is required
		// it is considered idiomatic
		goto cleanup;
	}

	// check whether the run was successful
	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch block scan kernel");

	// get the duration it took for the kernels to execute
	err = hipEventElapsedTime(&d_msecs, d_start, d_stop);
	CUDA_ERROR(err, "Failed to get elapsed time");

	// print the time elapsed
	printf(
			"Full block with bank avoidance scan with %d elements took = %.f5mSecs\n",
			len, d_msecs);

	// copy the result from the device back to the host
	err = hipMemcpy(h_OUT, d_OUT, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy array OUT from device to host");

	// -----------------------------------------------------------
	// 							CLEANUP
	// -----------------------------------------------------------

	cleanup:
	// Free device global memory
	CUDA_ERROR(hipFree(d_IN), "Failed to free device vector IN");
	CUDA_ERROR(hipFree(d_OUT), "Failed to free device vector OUT");
	CUDA_ERROR(hipFree(d_SUM_1), "Failed to free device vector SUM_1");
	CUDA_ERROR(hipFree(d_SUM_1_Scanned),"Failed to free device vector SUM_1_Scanned");
	CUDA_ERROR(hipFree(d_SUM_2), "Failed to free device vector SUM_2");
	CUDA_ERROR(hipFree(d_SUM_2_Scanned), "Failed to free device vector SUM_2_Scanned");

	// Clean up the Device timer event objects
	hipEventDestroy(d_start);
	hipEventDestroy(d_stop);

	// Reset the device and exit
	err = hipDeviceReset();
	CUDA_ERROR(err, "Failed to reset the device");

}

__global__
void block_scan_full(int *g_idata, int *g_odata, int n, int *SUM,
		int add_last) {
	__shared__ int temp[BLOCK_SIZE << 1];  // allocated on invocation

	int thid = threadIdx.x;
	int blockId = blockDim.x * blockIdx.x << 1;
	int offset = 0;
	int last = 0;

	if (blockId + (thid << 1) < n)
		temp[thid << 1] = g_idata[blockId + (thid << 1)]; // load input into shared memory
	if (blockId + (thid << 1) + 1 < n)
		temp[(thid << 1) + 1] = g_idata[blockId + (thid << 1) + 1];

	if (add_last && thid == BLOCK_SIZE - 1) // save the last element for later
		last = temp[(thid << 1) + 1];

	for (int d = BLOCK_SIZE /*n >> 1*/; d > 0; d >>= 1) // build sum in place up the tree
			{
		__syncthreads();
		if (thid < d) {
			int ai = (((thid << 1) + 1) << offset) - 1;
			int bi = (((thid << 1) + 2) << offset) - 1;
			temp[bi] += temp[ai];
		}
		offset++;
	}

	if (thid == 0) {
		temp[(BLOCK_SIZE << 1) - 1] = 0;
	} // clear the last element

	for (int d = 1; d < (BLOCK_SIZE << 1); d <<= 1) // traverse down tree & build scan
			{
		offset--;
		__syncthreads();
		if (thid < d) {
			int ai = (((thid << 1) + 1) << offset) - 1;
			int bi = (((thid << 1) + 2) << offset) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();
	if (add_last && thid == BLOCK_SIZE - 1) // save the last element for later
		SUM[blockIdx.x] = temp[(thid << 1) + 1] + last;
	if (blockId + (thid << 1) < n)
		g_odata[blockId + (thid << 1)] = temp[thid << 1]; // write results to device memory
	if (blockId + (thid << 1) + 1 < n)
		g_odata[blockId + (thid << 1) + 1] = temp[(thid << 1) + 1];
}

__host__
void full_block_scan(int *h_IN, int *h_OUT, int len) {

	// -----------------------------------------------------------
	// 							INITIALIZATION
	// -----------------------------------------------------------

	// error code to check return values for CUDA class
	hipError_t err = hipSuccess;

	// size to allocate for the vectors
	size_t size = len * sizeof(int);

	// create device timer
	hipEvent_t d_start, d_stop;
	float d_msecs;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);

	// allocate memory for all the possible vectors needed for the execution
	int *d_IN = NULL;
	err = hipMalloc((void **) &d_IN, size);
	CUDA_ERROR(err, "Failed to allocate device vector IN");

	int *d_OUT = NULL;
	err = hipMalloc((void**) &d_OUT, size);
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_1 = NULL;
	err = hipMalloc((void**) &d_SUM_1,
			(1 + ((len - 1) / (BLOCK_SIZE * 2))) * sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_1_Scanned = NULL;
	err = hipMalloc((void**) &d_SUM_1_Scanned,
			(1 + ((len - 1) / (BLOCK_SIZE * 2))) * sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_2 = NULL;
	err = hipMalloc((void**) &d_SUM_2, (BLOCK_SIZE << 1) * sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_2_Scanned = NULL;
	err = hipMalloc((void**) &d_SUM_2_Scanned,
			(BLOCK_SIZE << 1) * sizeof(int));
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	// copy the memory from the host to the device
	err = hipMemcpy(d_IN, h_IN, size, hipMemcpyHostToDevice);
	CUDA_ERROR(err, "Failed to copy array IN from host to device");

	// size of the grid for each level
	int blocksPerGridLevel1 = 1 + ((len - 1) / (BLOCK_SIZE * 2));
	int blocksPerGridLevel2 = 1 + ceil(blocksPerGridLevel1 / (BLOCK_SIZE << 1));
	int blocksPerGridLevel3 = 1 + ceil(blocksPerGridLevel2 / (BLOCK_SIZE << 1));

	// -----------------------------------------------------------
	// 							EXECUTION
	// -----------------------------------------------------------

	// choosing the level on which to run the kernels
	// based on the size of the grids

	// if level one grid size is equal to 1 then single
	// LEVEL 1 is enough to scan the whole array
	if (blocksPerGridLevel1 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);

		// execute the actual kernel
		block_scan_full<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT, len,
		NULL, 0);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if level two grid size is equal to 1 then two (LEVEL 2)
	// scans are required to scan the whole array
	else if (blocksPerGridLevel2 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);

		// execute the actual kernels
		block_scan_full<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT, len,
				d_SUM_1, 1);
		block_scan_full<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1,
				d_SUM_1_Scanned, blocksPerGridLevel1, NULL, 0);
		add_to_block<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_OUT, len,
				d_SUM_1_Scanned);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if level 3 grid size is equal to 1 then three (LEVEL 3)
	// scans are required to scan the whole array
	else if (blocksPerGridLevel3 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);

		// execute the actual kernels
		block_scan_full<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT, len,
				d_SUM_1, 1);
		block_scan_full<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1,
				d_SUM_1_Scanned, blocksPerGridLevel1, d_SUM_2, 1);
		block_scan_full<<<1, BLOCK_SIZE>>>(d_SUM_2, d_SUM_2_Scanned,
				blocksPerGridLevel2, NULL, 0);
		add_to_block<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1_Scanned,
				blocksPerGridLevel1, d_SUM_2_Scanned);
		add_to_block<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_OUT, len,
				d_SUM_1_Scanned);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if none of the conditions above is met, it means that the array is too
	// large to be scanned in 3 level scan, therefore we print the error message and return
	else {
		fprintf(stderr,
				"The array size=%d is too large to be scanned with level 3 scan!\n",
				len);

		// using goto is discouraged, however, in such situations
		// where in the error conditions exit or cleanup is required
		// it is considered idiomatic
		goto cleanup;
	}

	// check whether the run was successful
	err = hipGetLastError();
	CUDA_ERROR(err, "Failed to launch block scan kernel");

	// get the duration it took for the kernels to execute
	err = hipEventElapsedTime(&d_msecs, d_start, d_stop);
	CUDA_ERROR(err, "Failed to get elapsed time");

	// print the time elapsed
	printf("Full block scan with %d elements took = %.f5mSecs\n", len, d_msecs);

	// copy the result from the device back to the host
	err = hipMemcpy(h_OUT, d_OUT, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy array OUT from device to host");

	// -----------------------------------------------------------
	// 							CLEANUP
	// -----------------------------------------------------------

	cleanup:
	// Free device global memory
	CUDA_ERROR(hipFree(d_IN), "Failed to free device vector IN");
	CUDA_ERROR(hipFree(d_OUT), "Failed to free device vector OUT");
	CUDA_ERROR(hipFree(d_SUM_1), "Failed to free device vector SUM_1");
	CUDA_ERROR(hipFree(d_SUM_1_Scanned),
			"Failed to free device vector SUM_1_Scanned");
	CUDA_ERROR(hipFree(d_SUM_2), "Failed to free device vector SUM_2");
	CUDA_ERROR(hipFree(d_SUM_2_Scanned),
			"Failed to free device vector SUM_2_Scanned");

	// Clean up the Device timer event objects
	hipEventDestroy(d_start);
	hipEventDestroy(d_stop);

	// Reset the device and exit
	err = hipDeviceReset();
	CUDA_ERROR(err, "Failed to reset the device");
}

/**
 * Host main routine
 */
int main(void) {

// error code to check return calues for CUDA calss
	hipError_t err = hipSuccess;

// create host stopwatch times
	StopWatchInterface * timer = NULL;
	sdkCreateTimer(&timer);
	double h_msecs;

// create device timer
//	hipEvent_t d_start, d_stop;
//	float d_msecs;
//	hipEventCreate(&d_start);
//	hipEventCreate(&d_stop);

// size of the array to add
	int numElements = 10000000;
	size_t size = numElements * sizeof(int);

// allocate the memory on the host for the arrays
	int *h_IN = (int *) malloc(size);
	int *h_OUT = (int *) malloc(size);
	int *h_OUT_CUDA = (int *) malloc(size);

// verify the host allocations
	if (h_IN == NULL || h_OUT == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

// initialise the host input to 1.0f
	for (int i = 0; i < numElements; i++) {
		h_IN[i] = 1; //rand() % 10;
	}

// sequential scan
	sdkStartTimer(&timer);
	sequential_scan(h_IN, h_OUT, numElements);
	sdkStopTimer(&timer);
	h_msecs = sdkGetTimerValue(&timer);
	printf("Sequential scan on host of %d elements took = %.f5mSecs\n",
			numElements, h_msecs);

// ACtual algorithm

// allocate memory for the device
//	int *d_IN = NULL;
//	err = hipMalloc((void **) &d_IN, size);
//	CUDA_ERROR(err, "Failed to allocate device vector IN");
//
//	int *d_OUT = NULL;
//	err = hipMalloc((void**) &d_OUT, size);
//	CUDA_ERROR(err, "Failed to allocate device vector OUT");

// copy the memory from host to device
//	err = hipMemcpy(d_IN, h_IN, size, hipMemcpyHostToDevice);
//	CUDA_ERROR(err, "Failed to copy array IN from host to device");
//	int blocksPerGridLevel1 = 1 + ((numElements - 1) / (BLOCK_SIZE * 2));
//
//	hipEventRecord(d_start, 0);
//	block_scan_full_BCAO<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
//			numElements,
//			NULL, 0);
//
//	hipEventRecord(d_stop, 0);
//	hipEventSynchronize(d_stop);
//
//	hipDeviceSynchronize();
//	err = hipGetLastError();
//	CUDA_ERROR(err, "Failed to launch block scan kernel");
//
//	err = hipEventElapsedTime(&d_msecs, d_start, d_stop);
//	CUDA_ERROR(err, "Failed to get elapsed time");
//
//	printf("Block scan with single thread of %d elements took = %.f5mSecs\n",
//			numElements, d_msecs);
//
//	err = hipMemcpy(h_OUT_CUDA, d_OUT, size, hipMemcpyDeviceToHost);
//	CUDA_ERROR(err, "Failed to copy array OUT from device to host");
//
//	//// cleanup
//	// Free device global memory
//	err = hipFree(d_IN);
//	CUDA_ERROR(err, "Failed to free device vector A");
//	err = hipFree(d_OUT);
//	CUDA_ERROR(err, "Failed to free device vector B");
//
//	compare_results(h_OUT, h_OUT_CUDA, numElements);
////
	full_block_scan_BCAO(h_IN, h_OUT_CUDA, numElements);

	compare_results(h_OUT, h_OUT_CUDA, numElements);

	full_block_scan(h_IN, h_OUT_CUDA, numElements);

	compare_results(h_OUT, h_OUT_CUDA, numElements);

// Free host memory
	free(h_IN);
	free(h_OUT);
	free(h_OUT_CUDA);

// Clean up the Host timer
	sdkDeleteTimer(&timer);

// Clean up the Device timer event objects
//	hipEventDestroy(d_start);
//	hipEventDestroy(d_stop);
//
//// Reset the device and exit
//	err = hipDeviceReset();
//	CUDA_ERROR(err, "Failed to reset the device");

	return 0;

}
